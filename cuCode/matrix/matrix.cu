#include "hipblas.h"

#include "matrix.hpp"
#include "cuda_helper.hpp"
#include "CUDATimer.h"

#define H2D(dst, src, dataSize) checkCUDAError(hipMemcpy(dst, src, dataSize, hipMemcpyHostToDevice))
#define D2H(dst, src, dataSize) checkCUDAError(hipMemcpy(dst, src, dataSize, hipMemcpyDeviceToHost))

static void initMatrix(const int rows, const int cols, float *matrix){
	for(size_t i = 0; i < rows*cols; ++i){
		matrix[i] = rand() / (float)RAND_MAX;
	}
}

void matrixSingle(int M, int K, size_t N, int device){

	float *matrixA = new float[M*K];
	float *matrixB = new float[K*N];
	float *matrixC = new float[M*N];

	initMatrix(M, K, matrixA);
	initMatrix(K, N, matrixB);
	initMatrix(M, N, matrixC);

	float *d_matrixA = NULL;
	float *d_matrixB = NULL;
	float *d_matrixC = NULL;
	checkCUDAError(hipSetDevice(device));
	checkCUDAError(hipMalloc((void**)&d_matrixA, sizeof(float)*M*K));
	checkCUDAError(hipMalloc((void**)&d_matrixB, sizeof(float)*K*N));
	checkCUDAError(hipMalloc((void**)&d_matrixC, sizeof(float)*M*N));
	H2D(d_matrixA, matrixA, sizeof(float)*M*K);
	H2D(d_matrixB, matrixB, sizeof(float)*K*N);
	H2D(d_matrixC, matrixC, sizeof(float)*M*N);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasOperation_t transa = HIPBLAS_OP_N;
	hipblasOperation_t transb = HIPBLAS_OP_N;

	const float alpha = 0.5f;
	const float beta  = 1.0f;
	CUDATimer timer;
	timer.start();
	hipblasStatus_t status = hipblasSgemm(handle, transa, transb, M, N, K, &alpha, d_matrixA,
			M, d_matrixB, K, &beta, d_matrixC, M);
	timer.stop();
	float gflops = 2.0f*M*N*K / ((float)1000*1000) / timer.getElapsedMilliSeconds();
	std::cout << "CUDA Device " << device << " use time: " <<
		timer.getElapsedMilliSeconds() << "ms " << "Gflops: " << gflops << "GB/s" << std::endl;

	hipblasDestroy(handle);

	checkCUDAError(hipFree(d_matrixA));
	checkCUDAError(hipFree(d_matrixB));
	checkCUDAError(hipFree(d_matrixC));
}

